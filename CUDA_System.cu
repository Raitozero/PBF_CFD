#include "hip/hip_runtime.h"
#include "CUDA_System.h"
#include <math.h>
#include <random>

CUDASystem::CUDASystem(unsigned numParticles, glm::vec3 bounds_max, std::string config):System(numParticles, bounds_max) {
  //Set the parameters
  Parameters par;
  par.particleCount = numParticles;
  par.maxNeighbors = maxNeighbors;
  par.gravity = make_float3(gravity.x, gravity.y, gravity.z);
  par.bounds_min = make_float3(bounds_min.x, bounds_min.y, bounds_min.z);
  par.bounds_max = make_float3(bounds_max.x, bounds_max.y, bounds_max.z);
  par.iterations = iterations;
  par.dt = dt;
  par.h = h;
  par.rest_density = rest_density;
  par.epsilon = epsilon;
  par.k = k;
  par.dist_from_bound = dist_from_bound;
  par.delta_q = delta_q;
  par.viscosity_const = viscosity_const;
  par.poly6_const = poly6_const;
  par.spiky_const = spiky_const;

  //Set the grid
  par.maxGridCount = maxNeighbors;
  par.gridX = int(ceil((bounds_max.x - bounds_min.x) / h));
  par.gridY = int(ceil((bounds_max.y - bounds_min.y) / h));
  par.gridZ = int(ceil((bounds_max.z - bounds_min.z) / h));

  hostParticlePos = (float3 *)malloc(numParticles * sizeof(float3));
  std::default_random_engine generator;

  if (config == "dam") {
    std::uniform_real_distribution<float> distributionX(bounds_min.x + 0.1, bounds_min.x + 40);
    std::uniform_real_distribution<float> distributionY(bounds_min.y + 0.1, bounds_max.y - 0.1);
    std::uniform_real_distribution<float> distributionZ(bounds_min.z + 0.1, bounds_max.z - 0.1);
    for (int i = 0; i < numParticles; i++) {
      hostParticlePos[i] = make_float3(distributionX(generator), distributionY(generator), distributionZ(generator));
    }
  } else if (config == "sphere") {
    float r = std::min(std::min(bounds_max.x - bounds_min.x, bounds_max.y - bounds_min.y), bounds_max.z - bounds_min.z) / 2.0;
    float3 offset = make_float3((bounds_max.x - bounds_min.x) / 2.0, (bounds_max.y - bounds_min.y) / 2.0, (bounds_max.z - bounds_min.z) / 2.0);
    std::uniform_real_distribution<float> distributionR(-r, r);
    float x, y, z;
    for (int i = 0; i < numParticles; i++) {
      do {
        x = distributionR(generator);
        y = distributionR(generator);
        z = distributionR(generator);
      } while (x * x + y * y + z * z >= r * r);
      hostParticlePos[i] = make_float3(x, y, z) + offset;
    }
  } else {
    std::uniform_real_distribution<float> distribution(bounds_min.x + 5, bounds_max.x - 5);
    for (int i = 0; i < numParticles; i++) {
      hostParticlePos[i] =
          make_float3(distribution(generator), distribution(generator),
                      distribution(generator));
    }
  }

  gridSize = par.gridX * par.gridY * par.gridZ;

  cudaCheck(hipMalloc((void **)&particlePos, numParticles * sizeof(float3)));
  cudaCheck(hipMalloc((void **)&particleVel, numParticles * sizeof(float3)));
  cudaCheck(hipMalloc((void **)&next_position, numParticles * sizeof(float3)));
  cudaCheck(hipMalloc((void **)&particleLambda, numParticles * sizeof(float)));
  cudaCheck(hipMalloc((void **)&neighborCounts, numParticles * sizeof(int)));
  cudaCheck(hipMalloc((void **)&neighbors, numParticles * maxNeighbors * sizeof(int)));
  cudaCheck(hipMalloc((void **)&gridCount, gridSize * sizeof(int)));
  cudaCheck(hipMalloc((void **)&grid, gridSize * par.maxGridCount * sizeof(int)));

  cudaCheck(hipMemset(particlePos, 0, numParticles * sizeof(float3)));
  cudaCheck(hipMemset(particleVel, 0, numParticles * sizeof(float3)));
  cudaCheck(hipMemset(next_position, 0, numParticles * sizeof(float3)));
  cudaCheck(hipMemset(particleLambda, 0, numParticles * sizeof(float)));
  cudaCheck(hipMemset(neighborCounts, 0, numParticles * sizeof(int)));
  cudaCheck(hipMemset(neighbors, 0, numParticles * maxNeighbors * sizeof(int)));
  cudaCheck(hipMemset(gridCount, 0, gridSize * sizeof(int)));
  cudaCheck(hipMemset(grid, 0, gridSize * par.maxGridCount * sizeof(int)));

  cudaCheck(hipMemcpy(particlePos, hostParticlePos, numParticles * sizeof(float3), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(next_position, hostParticlePos, numParticles * sizeof(float3), hipMemcpyHostToDevice));
  initialize(&par);
}

CUDASystem::~CUDASystem() {
  cudaCheck(hipFree(particlePos));
  cudaCheck(hipFree(particleVel));
  cudaCheck(hipFree(next_position));
  cudaCheck(hipFree(particleLambda));
  cudaCheck(hipFree(neighborCounts));
  cudaCheck(hipFree(neighbors));
  cudaCheck(hipFree(gridCount));
  cudaCheck(hipFree(grid));
  free(hostParticlePos);
}

float *CUDASystem::getParticlePos() {
#ifdef DEVICE_RENDER
  return (float *)particlePos;

#else
  cudaCheck(hipMemcpy(hostParticlePos, particlePos,
                       numParticles * sizeof(float3), hipMemcpyDeviceToHost));
  return &hostParticlePos[0].x;

#endif
}

void CUDASystem::step() {
  update(gridSize, numParticles, iterations, particleVel, next_position,
         particlePos, neighborCounts, neighbors, gridCount, grid,
         particleLambda);
}
